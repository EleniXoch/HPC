#include "hip/hip_runtime.h"
/*
* This sample implements a separable convolution
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>


unsigned int filter_radius = 32;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.5 
#define PICS_SIZE 1024
#define PICS_SIZE_PADDED (PICS_SIZE + 2 * filter_radius)
// #define CPU
__constant__ double filterConst[513];

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter,
	int imageW, int imageH, int filterR) {

	int x, y, k;

	for (y = filterR; y < imageH + filterR; y++) {
		for (x = filterR; x < imageH + filterR; x++) {
			double sum = 0;

			for (k = -filterR; k <= filterR; k++) {
				int d = x + k;

				sum += h_Src[y * (imageW + 2 * filterR) + d] * h_Filter[filterR - k];

			}
			h_Dst[y * (imageW + 2 * filterR) + x] = sum;
		}

	}

}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter,
	int imageW, int imageH, int filterR) {

	int x, y, k;

	for (y = filterR; y < imageH + filterR; y++) {
		for (x = filterR; x < imageH + filterR; x++) {
			double sum = 0;

			for (k = -filterR; k <= filterR; k++) {
				int d = y + k;
				sum += h_Src[d * (imageW + 2 * filterR) + x] * h_Filter[filterR - k];
			}
			h_Dst[y * (imageW + 2 * filterR) + x] = sum;
		}
	}

}

__global__ void RowGPU(double *d_Dst, double * d_Src, int imageW, int imageH, int filterR) {


	register int k;
	int indexX = threadIdx.x + blockDim.x * blockIdx.x;
	int indexY = threadIdx.y + blockDim.y * blockIdx.y;
	int grid_width = gridDim.x * blockDim.x;
	register int idx = indexY * (grid_width + 2 * filterR) + indexX;
	register int padding_offset = (imageW + 2 * filterR)*filterR + filterR;
	register int d;

	register double sum = 0;
	for (k = -filterR; k <= filterR; k++) {
		d = indexX + k;
		sum += d_Src[indexY*(imageW + 2 * filterR) + d + padding_offset] * filterConst[filterR - k];
	}
	d_Dst[idx + padding_offset] = sum;

}

__global__ void ColGPU(double *d_Dst, const double  *d_Src, int imageW, int imageH, int filterR) {


	register int k;
	int indexX = threadIdx.x + blockDim.x * blockIdx.x;
	int indexY = threadIdx.y + blockDim.y * blockIdx.y;
	int grid_width = gridDim.x * blockDim.x;
	register int idx = indexY * (grid_width + 2 * filterR) + indexX;
	register int padding_offset = (imageW + 2 * filterR)*filterR + filterR;
	register int d;

	register double sum = 0;
	for (k = -filterR; k <= filterR; k++) {
		d = indexY + k;
		sum += d_Src[d*(imageW + 2 * filterR) + indexX + padding_offset] * filterConst[filterR - k];
	}
	d_Dst[idx + padding_offset] = sum;

}

void printArray(double *input, int size) {

	printf("The array is\n\n");
	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			printf("%lf ", input[i*size + j]);
		}
		printf("\n");
	}

}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {

	double
		*h_Filter,
		*h_Input,
		*h_Buffer,
		*h_OutputCPU,
		*h_OutputGPU,
		*h_OutputGPU_pic,
		*h_small_pic;

	double
		*d_Input,
		*d_Buffer,
		*d_OutputGPU_pic;


	int imageW;
	int imageH;
	int padding;
	int dim_padding;
	unsigned int i;

#ifdef CPU
	struct timespec tv1, tv2;
#endif
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	padding = 2 * filter_radius;

	printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
	scanf("%d", &imageW);
	imageH = imageW;
	dim_padding = imageW + padding;

	int runTimes = (imageH / PICS_SIZE) * (imageH / PICS_SIZE);
	int width = sqrt(runTimes);


	printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
	printf("Allocating and initializing host arrays...\n");
	// Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
	h_Filter = (double *)malloc(FILTER_LENGTH * sizeof(double));
	h_small_pic = (double *)calloc(PICS_SIZE_PADDED * PICS_SIZE_PADDED, sizeof(double));
	h_Input= (double *)calloc(dim_padding *dim_padding, sizeof(double));
	h_Buffer = (double *)calloc(dim_padding * dim_padding, sizeof(double));
	h_OutputCPU = (double *)calloc(dim_padding * dim_padding, sizeof(double));
	h_OutputGPU = (double *)calloc(dim_padding * dim_padding, sizeof(double));
	h_OutputGPU_pic = (double *)calloc(PICS_SIZE_PADDED * PICS_SIZE_PADDED, sizeof(double));

	// Allocate memory for the device
	hipError_t mallocErr2 = hipMalloc((void **)&d_Input, PICS_SIZE_PADDED * PICS_SIZE_PADDED * sizeof(double));
	hipError_t mallocErr3 = hipMalloc((void **)&d_Buffer, PICS_SIZE_PADDED * PICS_SIZE_PADDED * sizeof(double));
	hipError_t mallocErr4 = hipMalloc((void **)&d_OutputGPU_pic, PICS_SIZE_PADDED * PICS_SIZE_PADDED * sizeof(double));


	if (!h_Filter || !h_Input || !h_Buffer || !h_OutputCPU || !h_OutputGPU || !h_OutputGPU_pic) {
		fprintf(stderr, "malloc error\n");
		exit(1);
	}

	if (mallocErr2 != hipSuccess ||
		mallocErr3 != hipSuccess || mallocErr4 != hipSuccess) {
		fprintf(stderr, "hipMalloc error\n");
		exit(1);
	}



	srand(200);

	for (i = 0; i < FILTER_LENGTH; i++) {
		h_Filter[i] = (double)(rand() % 16);
	}

	//Initialize padding with zeros
	for (int i = 0; i < dim_padding; i++) {
		for (int j = 0; j < dim_padding; j++) {
			if (i < filter_radius || i > imageW + filter_radius - 1 || j < filter_radius || j > filter_radius + imageW - 1) {
				h_Input[i + j * dim_padding] = 0;
				h_Buffer[i + j * dim_padding] = 0;
				h_OutputCPU[i + j * dim_padding] = 0;
			}
			else {
				h_Input[i + j * dim_padding] = (double)rand() / ((double)RAND_MAX / 255) + (double)rand() / (double)RAND_MAX;
			}
		}
	}


	// To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
#ifdef CPU
	printf("CPU computation...\n");
	clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);
	convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
	convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
	clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);

	printf("%g\n",
		(double)(tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
		(double)(tv2.tv_sec - tv1.tv_sec));
#endif


	dim3 grid_dim;
	dim3 block_dim;

	if (PICS_SIZE > 32) {
		block_dim.x = 32;
		block_dim.y = 32;

		grid_dim.x = PICS_SIZE / block_dim.x;
		grid_dim.y = PICS_SIZE / block_dim.y;
	}
	else {
		grid_dim.x = 1;
		grid_dim.y = 1;

		block_dim.x = PICS_SIZE;
		block_dim.y = PICS_SIZE;
	}

	printf("GPU computation...\n");

	//Start measuring execution time of the two kernels
	hipEventRecord(start);
	hipMemcpyToSymbol(HIP_SYMBOL(filterConst), h_Filter, FILTER_LENGTH * sizeof(double));



	for (int id = 0; id < runTimes; id++) {
		//printf("(id%%width) %d (id / width) %d\n", (id%width), (id / width));
		for (int j = 0; j < PICS_SIZE_PADDED; j++) {
			for (int i = 0; i < PICS_SIZE_PADDED; i++) {
				h_small_pic[j*PICS_SIZE_PADDED + i] = h_Input[(i + (id%width)*PICS_SIZE) + (j + (id / width)*(PICS_SIZE))*(dim_padding)];
			}
		}


		//Copy small pic to d_Input
		hipMemcpy(d_Input, h_small_pic, PICS_SIZE_PADDED * PICS_SIZE_PADDED * sizeof(double), hipMemcpyHostToDevice);
		//Compute the rows and copy output to d_Buffer
		RowGPU << <grid_dim, block_dim >> > (d_Buffer, d_Input, PICS_SIZE, PICS_SIZE, filter_radius);

		hipMemcpy(h_OutputGPU_pic, d_Buffer, PICS_SIZE_PADDED * PICS_SIZE_PADDED * sizeof(double), hipMemcpyDeviceToHost);
		
		hipDeviceSynchronize();


		hipDeviceSynchronize();
		//transfer small to big array
		for (int j = filter_radius; j < PICS_SIZE + filter_radius; j++) {
			for (int i = filter_radius; i < PICS_SIZE + filter_radius; i++) {
				h_Buffer[(i + (id%width)*PICS_SIZE) + (j + (id / width)*(PICS_SIZE))*(dim_padding)] = h_OutputGPU_pic[j*PICS_SIZE_PADDED + i];
			}
		}


	}


	for (int id = 0; id < runTimes; id++) {

		for (int j = 0; j < PICS_SIZE_PADDED; j++) {
			for (int i = 0; i < PICS_SIZE_PADDED; i++) {
				h_small_pic[j*PICS_SIZE_PADDED + i] = h_Buffer[(i + (id%width)*PICS_SIZE) + (j + (id / width)*(PICS_SIZE))*(dim_padding)];
			}
		}


		//Copy small pic to d_Input
		hipMemcpy(d_Input, h_small_pic, PICS_SIZE_PADDED * PICS_SIZE_PADDED * sizeof(double), hipMemcpyHostToDevice);
		//Compute the rows and copy output to d_Buffer
		ColGPU << <grid_dim, block_dim >> > (d_Buffer, d_Input, PICS_SIZE, PICS_SIZE, filter_radius);
		hipMemcpy(h_OutputGPU_pic, d_Buffer, PICS_SIZE_PADDED * PICS_SIZE_PADDED * sizeof(double), hipMemcpyDeviceToHost);

		hipDeviceSynchronize();
		//transfer small to big array
		for (int j = filter_radius; j < PICS_SIZE + filter_radius; j++) {
			for (int i = filter_radius; i < PICS_SIZE + filter_radius; i++) {
				h_OutputGPU[(i + (id%width)*PICS_SIZE) + (j + (id / width)*(PICS_SIZE))*(dim_padding)] = h_OutputGPU_pic[j*PICS_SIZE_PADDED + i];
			}
		}


	}
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("kernel time in ms: %f\n", milliseconds);

#ifdef CPU
	for (i = 0; i < dim_padding * dim_padding; i++) {
		if (ABS(h_OutputGPU[i] - h_OutputCPU[i]) >= accuracy) {
			printf("error\n");
			break;
		}
	}
#endif

	// free all the allocated memory
	free(h_OutputCPU);
	free(h_Buffer);
	free(h_Input);
	free(h_Filter);
	free(h_OutputGPU);
	free(h_small_pic);
	free(h_OutputGPU_pic);

	//hipFree(d_Filter);
	hipFree(d_Input);
	hipFree(d_Buffer);
	hipFree(d_OutputGPU_pic);
	// Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
	hipDeviceReset();


	return 0;
}

