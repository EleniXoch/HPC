#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.h"

PGM_IMG contrast_enhancement_g(PGM_IMG img_in, int *hist, float *cdf_d, int *lut, unsigned char *out)
{
    PGM_IMG result;
    //int hist[256];
    
    //memset(hist, 0, 256);

    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));

    mempcpy(result.img, out, img_in.h*img_in.w*sizeof(unsigned char));
    

    //histogram(hist, img_in.img, img_in.h * img_in.w, 256);
    //histogram_equalization(result.img,img_in.img,hist,result.w*result.h, 256, cdf_d, lut);
    return result;
}
